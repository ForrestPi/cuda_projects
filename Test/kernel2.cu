#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hip/hip_runtime.h>  
#include <cctype>  
#include <cassert>  
#include <cstdio>  
#include <ctime>  
#include <cstdlib>


#define DATA_SIZE 1048576  
#define BLOCK_NUM 32  
#define THREAD_NUM 256  
#ifndef nullptr  
#define nullptr 0  
#endif  
   
using namespace std;  
   
////////////////////////���豸�����е��ں˺���/////////////////////////////  
__global__ static void Kernel_SquareSum( int* pIn, size_t* pDataSize,  
                               int*pOut, clock_t* pTime )  
{  
   // ����һ����̬����Ĺ���洢��  
   extern __shared__ int sharedData[];  
   
   const size_t computeSize =*pDataSize / THREAD_NUM;  
   const size_t tID = size_t(threadIdx.x );// �߳�  
   const size_t bID = size_t(blockIdx.x );// ��  
   
   int offset = 1;    // ��¼ÿ�������Ĳ���  
   int mask = 1;      // ѡ����ʵ��߳�  
   
   // ��ʼ��ʱ  
   if ( tID == 0 ) pTime[bID] =clock( );// ѡ������һ���߳̽��м�ʱ  
   
   // ִ�м���  
   for ( size_t i = bID * THREAD_NUM+ tID;  
      i < DATA_SIZE;  
      i += BLOCK_NUM * THREAD_NUM )  
   {  
      sharedData[tID] += pIn[i] * pIn[i];  
   }  
   
   // ͬ��һ�����е������߳�  
   __syncthreads( );  
   
   while ( offset < THREAD_NUM )  
   {  
      if ( ( tID & mask ) == 0 )  
      {  
         sharedData[tID] += sharedData[tID + offset];  
      }  
      offset += offset;     // ����һλ  
      mask = offset + mask; // �����һλ������λ  
   
      __syncthreads( );  
   }  
   
   if ( tID == 0 )// ����߳�IDΪ����ô������������¼ʱ��  
   {  
      pOut[bID] = sharedData[0];  
      pTime[bID + BLOCK_NUM] = clock( );  
   }  
}  
   
bool CUDA_SquareSum( int* pOut,clock_t* pTime,  
                int* pIn, size_t dataSize )  
{  
   assert( pIn != nullptr );  
   assert( pOut != nullptr );  
   
   int* pDevIn = nullptr;  
   int* pDevOut = nullptr;  
   size_t* pDevDataSize = nullptr;  
   clock_t* pDevTime = nullptr;  
   
   // 1�������豸  
   hipError_t cudaStatus = hipSetDevice( 0 );// ֻҪ������װ��Ӣΰ���Կ�����ô����óɹ�  
   if ( cudaStatus != hipSuccess )  
   {  
      fprintf( stderr, "����hipSetDevice()����ʧ�ܣ�" );  
      return false;  
   }  
   
   switch ( true)  
   {  
   default:  
      // 2�������Դ�ռ�  
      cudaStatus = hipMalloc( (void**)&pDevIn,dataSize * sizeof( int) );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMalloc()������ʼ���Կ�������ʱʧ�ܣ�" );  
         break;  
      }  
   
      cudaStatus = hipMalloc( (void**)&pDevOut,BLOCK_NUM * sizeof( int) );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMalloc()������ʼ���Կ��з���ֵʱʧ�ܣ�" );  
         break;  
      }  
   
      cudaStatus = hipMalloc( (void**)&pDevDataSize,sizeof( size_t ) );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMalloc()������ʼ���Կ������ݴ�Сʱʧ�ܣ�" );  
         break;  
      }  
   
      cudaStatus = hipMalloc( (void**)&pDevTime,BLOCK_NUM * 2 * sizeof( clock_t ) );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMalloc()������ʼ���Կ��кķ���ʱ����ʧ�ܣ�" );  
         break;  
      }  
   
      // 3���������������ݸ��Ƶ��Դ���  
      cudaStatus = hipMemcpy( pDevIn, pIn, dataSize * sizeof( int ),hipMemcpyHostToDevice );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMemcpy()������ʼ�����������������鵽�Կ�ʱʧ�ܣ�" );  
         break;  
      }  
   
      cudaStatus = hipMemcpy( pDevDataSize, &dataSize, sizeof( size_t ), hipMemcpyHostToDevice );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "����hipMemcpy()������ʼ�������������ݴ�С���Կ�ʱʧ�ܣ�" );  
         break;  
      }  
   
      // 4��ִ�г�����������ȴ��Կ�ִ�����  
      Kernel_SquareSum<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM *sizeof( int)>>>  
         ( pDevIn, pDevDataSize, pDevOut, pDevTime );  
   
      // 5����ѯ�ں˳�ʼ����ʱ���Ƿ����  
      cudaStatus = hipGetLastError( );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "�Կ�ִ�г���ʱʧ�ܣ�" );  
         break;  
      }  
   
      // 6�����ں�ͬ���ȴ�ִ�����  
      cudaStatus = hipDeviceSynchronize( );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "�����ں�ͬ���Ĺ����з������⣡" );  
         break;  
      }  
   
      // 7����ȡ����  
      cudaStatus = hipMemcpy( pOut, pDevOut, BLOCK_NUM * sizeof( int ),hipMemcpyDeviceToHost );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "�ڽ�������ݴ��Կ����Ƶ�����������ʧ�ܣ�" );  
         break;  
      }  
   
      cudaStatus = hipMemcpy( pTime, pDevTime, BLOCK_NUM * 2 * sizeof( clock_t ), hipMemcpyDeviceToHost );  
      if ( cudaStatus != hipSuccess)  
      {  
         fprintf( stderr, "�ڽ��ķ���ʱ���ݴ��Կ����Ƶ�����������ʧ�ܣ�" );  
         break;  
      }  
   
      hipFree( pDevIn );  
      hipFree( pDevOut );  
      hipFree( pDevDataSize );  
      hipFree( pDevTime );  
      return true;  
   }  
   
   hipFree( pDevIn );  
   hipFree( pDevOut );  
   hipFree( pDevDataSize );  
   hipFree( pDevTime );  
   return false;  
}  
   
void GenerateData( int* pData,size_t dataSize )// ��������  
{  
   assert( pData != nullptr );  
   for ( size_t i = 0; i <dataSize; i++ )  
   {  
      srand( i + 3 );  
      pData[i] = rand( ) % 100;  
   }  
}  
   
int main( int argc, char** argv )// �����������  
{  
   int* pData = nullptr;  
   int* pResult = nullptr;  
   clock_t* pTime = nullptr;  
   
   // ʹ��CUDA�ڴ����������host��  
   hipError_t cudaStatus = hipHostMalloc( &pData, DATA_SIZE * sizeof( int ) );  
   if ( cudaStatus != hipSuccess )  
   {  
      fprintf( stderr, "�������з�����Դʧ�ܣ�" );  
      return 1;  
   }  
   
   cudaStatus = hipHostMalloc( &pResult, BLOCK_NUM * sizeof( int ) );  
   if ( cudaStatus != hipSuccess )  
   {  
      fprintf( stderr, "�������з�����Դʧ�ܣ�" );  
      return 1;  
   }  
   
   cudaStatus = hipHostMalloc( &pTime, BLOCK_NUM * 2 * sizeof( clock_t ) );  
   if ( cudaStatus != hipSuccess )  
   {  
      fprintf( stderr, "�������з�����Դʧ�ܣ�" );  
      return 1;  
   }  
   
   GenerateData( pData, DATA_SIZE );// ͨ���������������  
   CUDA_SquareSum( pResult, pTime, pData, DATA_SIZE );// ִ��ƽ����  
   
   // ��CPU�н�����������  
   int totalResult=0;  
   for ( int i = 0; i < BLOCK_NUM; ++i )  
   {  
      totalResult += pResult[i];  
   }  
   
   // ����ִ�е�ʱ��  
   clock_t startTime = pTime[0];  
   clock_t endTime = pTime[BLOCK_NUM];  
   for ( int i = 0; i < BLOCK_NUM; ++i )  
   {  
      if ( startTime > pTime[i] )startTime = pTime[i];  
      if ( endTime < pTime[i +BLOCK_NUM] ) endTime = pTime[i + BLOCK_NUM];  
   }  
   clock_t elapsed = endTime - startTime;  
   
   
   // �ж��Ƿ����  
   char* pOverFlow = nullptr;  
   if ( totalResult < 0 )pOverFlow = "�������";  
   else pOverFlow = "";  
   
   // ��ʾ��׼����  
   printf( "��CUDA����ƽ���͵Ľ���ǣ�%d%s\n�ķ���ʱ��%d\n",  
      totalResult, pOverFlow, elapsed );  
   
   hipDeviceProp_t prop;  
   if ( hipGetDeviceProperties(&prop, 0 ) == hipSuccess )  
   {  
      float actualTime = float( elapsed ) / float(prop.clockRate );  
      printf( "ʵ��ִ��ʱ��Ϊ��%.2fms\n", actualTime );  
      printf( "����Ϊ��%.2fMB/s\n",  
         float( DATA_SIZE * sizeof( int )>> 20 ) * 1000.0f / actualTime );  
      printf( "GPU�豸�ͺţ�%s\n", prop.name );  
   }  
   
   hipHostFree( pData );  
   hipHostFree( pResult );  
   hipHostFree( pTime );  
   
   return 0;  
}  